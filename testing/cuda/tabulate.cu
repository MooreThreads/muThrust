#include "hip/hip_runtime.h"
/****************************************************************************
* This library contains code from thrust, thrust is licensed under the license
* below.
* Some files of thrust may have been modified by Moore Threads Technology Co.
* , Ltd
******************************************************************************/
#include <unittest/unittest.h>
#include <thrust/tabulate.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>


template<typename ExecutionPolicy, typename Iterator, typename Function>
__global__
void tabulate_kernel(ExecutionPolicy exec, Iterator first, Iterator last, Function f)
{
  thrust::tabulate(exec, first, last, f);
}


template<typename ExecutionPolicy>
void TestTabulateDevice(ExecutionPolicy exec)
{
  typedef thrust::device_vector<int> Vector;
  using namespace thrust::placeholders;
  typedef typename Vector::value_type T;
  
  Vector v(5);

  tabulate_kernel<<<1,1>>>(exec, v.begin(), v.end(), thrust::identity<T>());
  {
    musaError_t const err = musaDeviceSynchronize();
    ASSERT_EQUAL(musaSuccess, err);
  }

  ASSERT_EQUAL(v[0], 0);
  ASSERT_EQUAL(v[1], 1);
  ASSERT_EQUAL(v[2], 2);
  ASSERT_EQUAL(v[3], 3);
  ASSERT_EQUAL(v[4], 4);

  tabulate_kernel<<<1,1>>>(exec, v.begin(), v.end(), -_1);
  {
    musaError_t const err = musaDeviceSynchronize();
    ASSERT_EQUAL(musaSuccess, err);
  }

  ASSERT_EQUAL(v[0],  0);
  ASSERT_EQUAL(v[1], -1);
  ASSERT_EQUAL(v[2], -2);
  ASSERT_EQUAL(v[3], -3);
  ASSERT_EQUAL(v[4], -4);
  
  tabulate_kernel<<<1,1>>>(exec, v.begin(), v.end(), _1 * _1 * _1);
  {
    musaError_t const err = musaDeviceSynchronize();
    ASSERT_EQUAL(musaSuccess, err);
  }

  ASSERT_EQUAL(v[0], 0);
  ASSERT_EQUAL(v[1], 1);
  ASSERT_EQUAL(v[2], 8);
  ASSERT_EQUAL(v[3], 27);
  ASSERT_EQUAL(v[4], 64);
}

void TestTabulateDeviceSeq()
{
  TestTabulateDevice(thrust::seq);
}
DECLARE_UNITTEST(TestTabulateDeviceSeq);

void TestTabulateDeviceDevice()
{
  TestTabulateDevice(thrust::device);
}
DECLARE_UNITTEST(TestTabulateDeviceDevice);

void TestTabulateCudaStreams()
{
  using namespace thrust::placeholders;
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;
  
  Vector v(5);

  musaStream_t s;
  musaStreamCreate(&s);

  thrust::tabulate(thrust::musa::par.on(s), v.begin(), v.end(), thrust::identity<T>());
  musaStreamSynchronize(s);

  ASSERT_EQUAL(v[0], 0);
  ASSERT_EQUAL(v[1], 1);
  ASSERT_EQUAL(v[2], 2);
  ASSERT_EQUAL(v[3], 3);
  ASSERT_EQUAL(v[4], 4);

  thrust::tabulate(thrust::musa::par.on(s), v.begin(), v.end(), -_1);
  musaStreamSynchronize(s);

  ASSERT_EQUAL(v[0],  0);
  ASSERT_EQUAL(v[1], -1);
  ASSERT_EQUAL(v[2], -2);
  ASSERT_EQUAL(v[3], -3);
  ASSERT_EQUAL(v[4], -4);
  
  thrust::tabulate(thrust::musa::par.on(s), v.begin(), v.end(), _1 * _1 * _1);
  musaStreamSynchronize(s);

  ASSERT_EQUAL(v[0], 0);
  ASSERT_EQUAL(v[1], 1);
  ASSERT_EQUAL(v[2], 8);
  ASSERT_EQUAL(v[3], 27);
  ASSERT_EQUAL(v[4], 64);

  musaStreamSynchronize(s);
}
DECLARE_UNITTEST(TestTabulateCudaStreams);

