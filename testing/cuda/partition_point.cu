#include "hip/hip_runtime.h"
/****************************************************************************
* This library contains code from thrust, thrust is licensed under the license
* below.
* Some files of thrust may have been modified by Moore Threads Technology Co.
* , Ltd
******************************************************************************/
#include <unittest/unittest.h>
#include <thrust/partition.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>


template<typename ExecutionPolicy, typename Iterator1, typename Predicate, typename Iterator2>
__global__
void partition_point_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Predicate pred, Iterator2 result)
{
  *result = thrust::partition_point(exec, first, last, pred);
}


template<typename T>
struct is_even
{
  __host__ __device__
  bool operator()(T x) const { return ((int) x % 2) == 0; }
};


template<typename ExecutionPolicy>
void TestPartitionPointDevice(ExecutionPolicy exec)
{
  size_t n = 1000;
  thrust::device_vector<int> v = unittest::random_integers<int>(n);
  typedef typename thrust::device_vector<int>::iterator iterator;

  iterator ref = thrust::stable_partition(v.begin(), v.end(), is_even<int>());

  thrust::device_vector<iterator> result(1);
  partition_point_kernel<<<1,1>>>(exec, v.begin(), v.end(), is_even<int>(), result.begin());
  musaError_t const err = musaDeviceSynchronize();
  ASSERT_EQUAL(musaSuccess, err);

  ASSERT_EQUAL(ref - v.begin(), (iterator)result[0] - v.begin());
}


void TestPartitionPointDeviceSeq()
{
  TestPartitionPointDevice(thrust::seq);
}
DECLARE_UNITTEST(TestPartitionPointDeviceSeq);


void TestPartitionPointDeviceDevice()
{
  TestPartitionPointDevice(thrust::device);
}
DECLARE_UNITTEST(TestPartitionPointDeviceDevice);


void TestPartitionPointCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;
  typedef Vector::iterator Iterator;

  Vector v(4);
  v[0] = 1; v[1] = 1; v[2] = 1; v[3] = 0;

  Iterator first = v.begin();

  Iterator last = v.begin() + 4;
  Iterator ref = first + 3;

  musaStream_t s;
  musaStreamCreate(&s);

  ASSERT_EQUAL_QUIET(ref, thrust::partition_point(thrust::musa::par.on(s), first, last, thrust::identity<T>()));

  last = v.begin() + 3;
  ref = last;
  ASSERT_EQUAL_QUIET(ref, thrust::partition_point(thrust::musa::par.on(s), first, last, thrust::identity<T>()));

  musaStreamDestroy(s);
}
DECLARE_UNITTEST(TestPartitionPointCudaStreams);

