#include "hip/hip_runtime.h"
/****************************************************************************
* This library contains code from thrust, thrust is licensed under the license
* below.
* Some files of thrust may have been modified by Moore Threads Technology Co.
* , Ltd
******************************************************************************/
#include <unittest/unittest.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>


template<typename ExecutionPolicy, typename Iterator>
__global__
void sequence_kernel(ExecutionPolicy exec, Iterator first, Iterator last)
{
  thrust::sequence(exec, first, last);
}


template<typename ExecutionPolicy, typename Iterator, typename T>
__global__
void sequence_kernel(ExecutionPolicy exec, Iterator first, Iterator last, T init)
{
  thrust::sequence(exec, first, last, init);
}


template<typename ExecutionPolicy, typename Iterator, typename T>
__global__
void sequence_kernel(ExecutionPolicy exec, Iterator first, Iterator last, T init, T step)
{
  thrust::sequence(exec, first, last, init, step);
}


template<typename ExecutionPolicy>
void TestSequenceDevice(ExecutionPolicy exec)
{
  thrust::device_vector<int> v(5);
  
  sequence_kernel<<<1,1>>>(exec, v.begin(), v.end());
  {
    musaError_t const err = musaDeviceSynchronize();
    ASSERT_EQUAL(musaSuccess, err);
  }
 
  ASSERT_EQUAL(v[0], 0);
  ASSERT_EQUAL(v[1], 1);
  ASSERT_EQUAL(v[2], 2);
  ASSERT_EQUAL(v[3], 3);
  ASSERT_EQUAL(v[4], 4);
  
  sequence_kernel<<<1,1>>>(exec, v.begin(), v.end(), 10);
  {
    musaError_t const err = musaDeviceSynchronize();
    ASSERT_EQUAL(musaSuccess, err);
  }
  
  ASSERT_EQUAL(v[0], 10);
  ASSERT_EQUAL(v[1], 11);
  ASSERT_EQUAL(v[2], 12);
  ASSERT_EQUAL(v[3], 13);
  ASSERT_EQUAL(v[4], 14);
  
  sequence_kernel<<<1,1>>>(exec, v.begin(), v.end(), 10, 2);
  {
    musaError_t const err = musaDeviceSynchronize();
    ASSERT_EQUAL(musaSuccess, err);
  }
  
  ASSERT_EQUAL(v[0], 10);
  ASSERT_EQUAL(v[1], 12);
  ASSERT_EQUAL(v[2], 14);
  ASSERT_EQUAL(v[3], 16);
  ASSERT_EQUAL(v[4], 18);
}

void TestSequenceDeviceSeq()
{
  TestSequenceDevice(thrust::seq);
}
DECLARE_UNITTEST(TestSequenceDeviceSeq);

void TestSequenceDeviceDevice()
{
  TestSequenceDevice(thrust::device);
}
DECLARE_UNITTEST(TestSequenceDeviceDevice);

void TestSequenceCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  
  Vector v(5);

  musaStream_t s;
  musaStreamCreate(&s);

  thrust::sequence(thrust::musa::par.on(s), v.begin(), v.end());
  musaStreamSynchronize(s);

  ASSERT_EQUAL(v[0], 0);
  ASSERT_EQUAL(v[1], 1);
  ASSERT_EQUAL(v[2], 2);
  ASSERT_EQUAL(v[3], 3);
  ASSERT_EQUAL(v[4], 4);

  thrust::sequence(thrust::musa::par.on(s), v.begin(), v.end(), 10);
  musaStreamSynchronize(s);

  ASSERT_EQUAL(v[0], 10);
  ASSERT_EQUAL(v[1], 11);
  ASSERT_EQUAL(v[2], 12);
  ASSERT_EQUAL(v[3], 13);
  ASSERT_EQUAL(v[4], 14);
  
  thrust::sequence(thrust::musa::par.on(s), v.begin(), v.end(), 10, 2);
  musaStreamSynchronize(s);

  ASSERT_EQUAL(v[0], 10);
  ASSERT_EQUAL(v[1], 12);
  ASSERT_EQUAL(v[2], 14);
  ASSERT_EQUAL(v[3], 16);
  ASSERT_EQUAL(v[4], 18);

  musaStreamDestroy(s);
}
DECLARE_UNITTEST(TestSequenceCudaStreams);

