#include "hip/hip_runtime.h"
/****************************************************************************
* This library contains code from thrust, thrust is licensed under the license
* below.
* Some files of thrust may have been modified by Moore Threads Technology Co.
* , Ltd
******************************************************************************/
#include <unittest/unittest.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>


template<typename ExecutionPolicy, typename Iterator, typename T, typename Iterator2>
__global__
void reduce_kernel(ExecutionPolicy exec, Iterator first, Iterator last, T init, Iterator2 result)
{
  *result = thrust::reduce(exec, first, last, init);
}


template<typename T, typename ExecutionPolicy>
void TestReduceDevice(ExecutionPolicy exec, const size_t n)
{
  thrust::host_vector<T>   h_data = unittest::random_integers<T>(n);
  thrust::device_vector<T> d_data = h_data;
  
  thrust::device_vector<T> d_result(1);
  
  T init = 13;
  
  T h_result = thrust::reduce(h_data.begin(), h_data.end(), init);
  
  reduce_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), init, d_result.begin());
  musaError_t const err = musaDeviceSynchronize();
  ASSERT_EQUAL(musaSuccess, err);
  
  ASSERT_EQUAL(h_result, d_result[0]);
}


template<typename T>
struct TestReduceDeviceSeq
{
  void operator()(const size_t n)
  {
    TestReduceDevice<T>(thrust::seq, n);
  }
};
VariableUnitTest<TestReduceDeviceSeq, IntegralTypes> TestReduceDeviceSeqInstance;


template<typename T>
struct TestReduceDeviceDevice
{
  void operator()(const size_t n)
  {
    TestReduceDevice<T>(thrust::device, n);
  }
};
VariableUnitTest<TestReduceDeviceDevice, IntegralTypes> TestReduceDeviceDeviceInstance;


void TestReduceCudaStreams()
{
  typedef thrust::device_vector<int> Vector;

  Vector v(3);
  v[0] = 1; v[1] = -2; v[2] = 3;

  musaStream_t s;
  musaStreamCreate(&s);

  // no initializer
  ASSERT_EQUAL(thrust::reduce(thrust::musa::par.on(s), v.begin(), v.end()), 2);

  // with initializer
  ASSERT_EQUAL(thrust::reduce(thrust::musa::par.on(s), v.begin(), v.end(), 10), 12);

  musaStreamDestroy(s);
}
DECLARE_UNITTEST(TestReduceCudaStreams);

