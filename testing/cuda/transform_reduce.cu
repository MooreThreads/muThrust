#include "hip/hip_runtime.h"
/****************************************************************************
* This library contains code from thrust, thrust is licensed under the license
* below.
* Some files of thrust may have been modified by Moore Threads Technology Co.
* , Ltd
******************************************************************************/
#include <unittest/unittest.h>
#include <thrust/transform_reduce.h>
#include <thrust/execution_policy.h>


template<typename ExecutionPolicy, typename Iterator1, typename Function1, typename T, typename Function2, typename Iterator2>
__global__
void transform_reduce_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Function1 f1, T init, Function2 f2, Iterator2 result)
{
  *result = thrust::transform_reduce(exec, first, last, f1, init, f2);
}


template<typename ExecutionPolicy>
void TestTransformReduceDevice(ExecutionPolicy exec)
{
  typedef thrust::device_vector<int> Vector;
  typedef typename Vector::value_type T;
  
  Vector data(3);
  data[0] = 1; data[1] = -2; data[2] = 3;
  
  T init = 10;

  thrust::device_vector<T> result(1);

  transform_reduce_kernel<<<1,1>>>(exec, data.begin(), data.end(), thrust::negate<T>(), init, thrust::plus<T>(), result.begin());
  musaError_t const err = musaDeviceSynchronize();
  ASSERT_EQUAL(musaSuccess, err);
  
  ASSERT_EQUAL(8, (T)result[0]);
}


void TestTransformReduceDeviceSeq()
{
  TestTransformReduceDevice(thrust::seq);
}
DECLARE_UNITTEST(TestTransformReduceDeviceSeq);


void TestTransformReduceDeviceDevice()
{
  TestTransformReduceDevice(thrust::device);
}
DECLARE_UNITTEST(TestTransformReduceDeviceDevice);


void TestTransformReduceCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef Vector::value_type T;
  
  Vector data(3);
  data[0] = 1; data[1] = -2; data[2] = 3;
  
  T init = 10;

  musaStream_t s;
  musaStreamCreate(&s);

  T result = thrust::transform_reduce(thrust::musa::par.on(s), data.begin(), data.end(), thrust::negate<T>(), init, thrust::plus<T>());
  musaStreamSynchronize(s);
  
  ASSERT_EQUAL(8, result);

  musaStreamDestroy(s);
}
DECLARE_UNITTEST(TestTransformReduceCudaStreams);

